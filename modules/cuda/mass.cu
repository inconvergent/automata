#include "hip/hip_runtime.h"
#define THREADS _THREADS_

__device__ void _calculate_centre_of_mass(
    const unsigned int grid_size,
    const unsigned int i,
    const unsigned int j,
    const bool *grid,
    const unsigned int influence_rad,
    float *massx,
    float *massy,
    int *neigh
    ){
  const float one = 1.0f/(float)grid_size;

  int count = 0;

  float mx = 0.0f;
  float my = 0.0f;
  float nrm = 0.0f;

  const float x = i*one;
  const float y = j*one;
  float dx = 0;
  float dy = 0;

  const float rad = pow(one*(float)influence_rad, 2.0f);

  int k;
  for (unsigned int a=max(i-influence_rad,0);a<min(i+influence_rad+1,grid_size);a++){
    for (unsigned int b=max(j-influence_rad,0);b<min(j+influence_rad+1,grid_size);b++){
      k = a*grid_size+b;
      if (grid[k]){
        dx = x-a*one;
        dy = y-b*one;
        nrm = dx*dx+dy*dy;
        if (nrm>rad){
          continue;
        }
        mx += a*one;
        my += b*one;
        count += 1;
      }
    }
  }

  k = i*grid_size+j;
  neigh[k] = count;

  if (count>0){
    mx /= (float)count;
    my /= (float)count;

    mx = x-mx;
    my = y-my;

    nrm = mx*mx+my*my;
    if (nrm>0.0f){
      nrm = sqrt(nrm);
      massx[k] = mx/nrm;
      massy[k] = my/nrm;
    }
    else{
      massx[k] = 0.0f;
      massy[k] = 0.0f;
    }
  }
  else{
    massx[k] = 0.0f;
    massy[k] = 0.0f;
  }

  return;
}

__device__ void _count_connected(
    const unsigned int grid_size,
    const int i,
    const int j,
    const bool *grid,
    int *connected
    ){

  int k = i*grid_size+j;

  int count = 0;

  for (int a=max(i-1,0);a<min(i+2,grid_size);a++){
    for (int b=max(j-1,0);b<min(j+2,grid_size);b++){
      k = a*grid_size+b;
      if (grid[k]){
        count += 1;
      }
    }
  }
  k = i*grid_size+j;
  connected[k] = count;

  return;
}

__global__ void mass(
    const int n,
    const unsigned int grid_size,
    const bool *grid,
    const int influence_rad,
    float *massx,
    float *massy,
    int *neigh,
    int *connected
    ){
  const unsigned int ij = blockIdx.x*THREADS + threadIdx.x;
  const unsigned int i = ij/grid_size;
  const unsigned int j = ij%grid_size;

  if (ij>=n){
    return;
  }

  _calculate_centre_of_mass(grid_size, i, j, grid, influence_rad, massx, massy, neigh);
  _count_connected(grid_size, i, j, grid, connected);

}
